#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
#include "utils.h"

const int BLOCKSIZE = 128;
const int NUMBLOCKS = 1000;
// set this to 1 or 2 for debugging

const int N 	    = BLOCKSIZE*NUMBLOCKS;

/* 
 * TODO: modify the foo and bar kernels to use tiling: 
 * 		 - copy the input data to shared memory
 *		 - perform the computation there
 *	     - copy the result back to global memory
 *		 - assume thread blocks of 128 threads
 *		 - handle intra-block boundaries correctly
 * You can ignore boundary conditions (we ignore the first 2 and last
 * 2 elements)
 */

__global__ void foo(float out[], float A[], float B[], float C[], float D[], float E[]){  
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
}

__global__ void foo_tile(float out[], float A[], float B[], float C[], float D[], float E[]){  
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int x = threadIdx.x;
  __shared__ float tile [128];

  // Copy input to tile

  tile [x] = A[i] + B[i] + C[i] + D[i] + E[i];
  __syncthreads ();
  
  out[i] = tile[x] / 5.0f;
}

__global__ void bar(float out[], float in[]) 
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
}

__global__ void bar_tile(float out[], float in[]) 
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int x = threadIdx.x;
  __shared__ float tile [128 + 4];

  // Copy input to tile

  tile [x + 2] = in[i];
  if (x == 0) {
    tile[x] = in[i-2];
    tile[x+1] = in[i-1];
  }
  else if (x == blockDim.x - 1) {
    tile[x + 3] = in[i+1];
    tile[x + 4] = in[i+2];
  }

  __syncthreads();
  
  //out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
  out[i] = (tile[x] + tile[x + 1] + tile[x + 2] + tile[x + 3] + tile[x + 4]) / 5.0f;

}

void cpuFoo(float out[], float A[], float B[], float C[], float D[], float E[])
{
  for (int i=0; i<N; i++)
    {
      out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
    }
}

void cpuBar(float out[], float in[])
{
  // ignore the boundaries
  for (int i=2; i<N-2; i++)
    {
      out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
    }
}

int main(int argc, char **argv)
{
  // declare and fill input arrays for foo() and bar()
  float fooA[N], fooB[N], fooC[N], fooD[N], fooE[N], barIn[N];
  for (int i=0; i<N; i++) 
    {
      fooA[i] = i; 
      fooB[i] = i+1;
      fooC[i] = i+2;
      fooD[i] = i+3;
      fooE[i] = i+4;
      barIn[i] = 2*i; 
    }
  // device arrays
  int numBytes = N * sizeof(float);
  float *d_fooA;	 	hipMalloc(&d_fooA, numBytes);
  float *d_fooB; 		hipMalloc(&d_fooB, numBytes);
  float *d_fooC;	 	hipMalloc(&d_fooC, numBytes);
  float *d_fooD; 		hipMalloc(&d_fooD, numBytes);
  float *d_fooE; 		hipMalloc(&d_fooE, numBytes);
  float *d_barIn; 	hipMalloc(&d_barIn, numBytes);
  hipMemcpy(d_fooA, fooA, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_fooB, fooB, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_fooC, fooC, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_fooD, fooD, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_fooE, fooE, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_barIn, barIn, numBytes, hipMemcpyHostToDevice);	
  
  // output arrays for host and device
  float fooOut[N], barOut[N], *d_fooOut, *d_barOut;
  hipMalloc(&d_fooOut, numBytes);
  hipMalloc(&d_barOut, numBytes);
  
  // declare and compute reference solutions
  float ref_fooOut[N], ref_barOut[N];
  GpuTimer fooCpuTimer, barCpuTimer;
  
  fooCpuTimer.Start();
  cpuFoo(ref_fooOut, fooA, fooB, fooC, fooD, fooE);
  fooCpuTimer.Stop();
  
  barCpuTimer.Start();
  cpuBar(ref_barOut, barIn);
  barCpuTimer.Stop();
  
  // launch and time foo and bar
  GpuTimer fooTimer, barTimer;
  fooTimer.Start();
  foo<<<N/BLOCKSIZE, BLOCKSIZE>>>
    (d_fooOut, d_fooA, d_fooB, d_fooC, d_fooD, d_fooE);
  fooTimer.Stop();

  hipMemcpy(fooOut, d_fooOut, numBytes, hipMemcpyDeviceToHost);
  printf("foo<<<>>>(): %g ms elapsed. Verifying solution...",
	 fooTimer.Elapsed());
  compareArrays(ref_fooOut, fooOut, N);

  fooTimer.Start();
  foo_tile<<<N/BLOCKSIZE, BLOCKSIZE>>>
    (d_fooOut, d_fooA, d_fooB, d_fooC, d_fooD, d_fooE);
  fooTimer.Stop();

  hipMemcpy(fooOut, d_fooOut, numBytes, hipMemcpyDeviceToHost);
  printf("foo_tile<<<>>>(): %g ms elapsed. Verifying solution...",
	 fooTimer.Elapsed());
  compareArrays(ref_fooOut, fooOut, N);
  
  barTimer.Start();
  bar<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_barOut, d_barIn);
  barTimer.Stop();
  
  hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost);
  printf("bar<<<>>>(): %g ms elapsed. Verifying solution...",
	 barTimer.Elapsed());
  compareArrays(ref_barOut, barOut, N);

  barTimer.Start();
  bar_tile<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_barOut, d_barIn);
  barTimer.Stop();
  
  hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost);
  printf("bar_tile<<<>>>(): %g ms elapsed. Verifying solution...",
	 barTimer.Elapsed());
  compareArrays(ref_barOut, barOut, N);
  
  printf("fooCpu(): %g ms elapsed.\n", fooCpuTimer.Elapsed());
  printf("barCpu(): %g ms elapsed.\n", barCpuTimer.Elapsed());
}
