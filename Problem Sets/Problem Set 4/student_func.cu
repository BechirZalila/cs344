#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__ void histo_kernel(unsigned int * d_out,
			     unsigned int* const d_in,
			     unsigned int shift,
			     const unsigned int numElems)
{
  // Only 2 bins (one for zeroes and on for ones)
  
  unsigned int mask = 1 << shift;
  
  int myId = threadIdx.x + blockDim.x * blockIdx.x;

  if (myId >= numElems)
    return;

  int bin = (d_in[myId] & mask) >> shift;
  atomicAdd(&d_out[bin], 1);
}

// Blelloch Scan - described in lecture
__global__ void sumscan_kernel(unsigned int * d_in,
			       const size_t numBins,
			       const unsigned int numElems)
{
  int myId = threadIdx.x;
  
  if (myId >= numElems)
    return;

  extern __shared__ float sdata[]; // Allocated on kernel call
  
  sdata[myId] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!

  for (int d = 1; d < numBins; d *= 2) {
    if (myId >= d) {
      sdata[myId] += sdata[myId - d];
    }
    __syncthreads();
  }
  
  if (myId == 0)
    d_in[0] = 0;
  else
    d_in[myId] = sdata[myId - 1]; //inclusive->exclusive
}

__global__ void makescan_kernel(unsigned int * d_in,
				unsigned int *d_scan,
				unsigned int shift,
				const unsigned int numElems)
{
  unsigned int mask = 1 << shift;
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  
  if (myId >= numElems)
    return;

  d_scan[myId] = ((d_in[myId] & mask) >> shift) ? 0 : 1;
}

__global__ void move_kernel(unsigned int* const d_inputVals,
			    unsigned int* const d_inputPos,
			    unsigned int* const d_outputVals,
			    unsigned int* const d_outputPos,
			    const unsigned int numElems,
			    unsigned int* const d_histogram,
			    unsigned int* const d_scaned,
			    unsigned int shift)
{
  unsigned int mask = 1 << shift;
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  
  if (myId >= numElems)
    return;

  // Algorithm described in 7.4 of http://wykvictor.github.io/2016/04/03/Cuda-2.html 

  int des_id = 0;

  if ((d_inputVals[myId] & mask) >> shift) {
    des_id = myId + d_histogram[1] - d_scaned[myId];
  } else {
    des_id = d_scaned[myId];
  }
  
  d_outputVals[des_id] = d_inputVals[myId];
  d_outputPos[des_id] = d_inputPos[myId];
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  
}
